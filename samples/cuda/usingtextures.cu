#include "hip/hip_runtime.h"

__global__ void kernel(hipTextureObject_t tex, float* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float value = tex2D<float>(tex, x, y);  // Получаем значение из текстурной памяти
        output[y * width + x] = value * 2.0f;  // Пример работы с данными
    }
}



int main()
{

hipTextureObject_t tex;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
hipArray_t cuArray;

// Allocate texture memory
hipMallocArray(&cuArray, &channelDesc, width, height);

// Copy data to texture
hipMemcpyToArray(cuArray, 0, 0, h_data, width * height * sizeof(float), hipMemcpyHostToDevice);

// Tesure memory description
hipResourceDesc resDesc = {};
resDesc.resType = hipResourceTypeArray;
resDesc.res.array.array = cuArray;

hipTextureDesc texDesc = {};
texDesc.addressMode[0] = hipAddressModeClamp;
texDesc.addressMode[1] = hipAddressModeClamp;
texDesc.filterMode = hipFilterModePoint;
texDesc.readMode = hipReadModeElementType;
texDesc.normalizedCoords = false;


hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);


kernel<<<>>>

hipDestroyTextureObject(tex);
hipFreeArray(cuArray);



}